#include "hip/hip_runtime.h"
__global__ void gemm_smem_cacheblocking(float* data0, float* data1, float* data2,
                                        int M, int N, int K, int CHUNKSIZE) {
    __shared__ float As[CHUNKSIZE * CHUNKSIZE];
    __shared__ float Bs[CHUNKSIZE * CHUNKSIZE];
    int cRow = blockIdx.x;
    int cCol = blockIdx.y;
    int tCol = threadIdx.x % CHUNKSIZE;
    int tRow = threadIdx.x / CHUNKSIZE;
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    int b = threadIdx.x + blockDim.x * blockIdx.x;
    int c = (1 * threadIdx.x) + blockDim.x * blockIdx.x;
    int d = (0 + threadIdx.x) + blockDim.x * blockIdx.x;
    int e = (5 * threadIdx.x) + blockDim.x * blockIdx.x;
    int f = (2 * 1 * threadIdx.x) + blockDim.x * 1 * blockIdx.x;
    int g = (2 * 5 * 1 * blockIdx.x) + blockDim.x * 1 * threadIdx.x;

    data0 = data0 + cRow * CHUNKSIZE * K;
    data1 = data1 + cCol * CHUNKSIZE;
    data2 = data2 + cRow * CHUNKSIZE * N + cCol * CHUNKSIZE;
    
    float tmp = 0.0;
    for (int bkIdx = 0; bkIdx < K; bkIdx = bkIdx + CHUNKSIZE) {
        data0_s[tRow * CHUNKSIZE + tCol] = data0[tRow * K + tCol];
        data1_s[tRow * CHUNKSIZE + tCol] = data1[tRow * N + tCol];
        data0 = data0 + CHUNKSIZE;
        data1 = data1 + CHUNKSIZE * N;
        for (int dotIdx = 0; dotIdx < CHUNKSIZE; dotIdx = dotIdx + 1) {
            tmp = tmp + data0_s[tRow * CHUNKSIZE + dotIdx] * data1_s[dotIdx * CHUNKSIZE + tCol];
        }
    }
    data2[tRow * N + tCol] = tmp;
}