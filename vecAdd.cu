
#include <hip/hip_runtime.h>
__global__ void vecAdd(int* A, int* B, int* C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    C[idx] = A[idx] + B[idx];
}